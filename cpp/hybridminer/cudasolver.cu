
 #include "cudasolver.h"

#include <assert.h>

#include <sstream>
#include <iomanip>
#include <stdio.h>

#include <iostream>
#include <string.h>
using namespace std;

//we will need this!



 #include "cuda_sha3.cu"



static const char* const ascii[] = {
  "00","01","02","03","04","05","06","07","08","09","0a","0b","0c","0d","0e","0f",
  "10","11","12","13","14","15","16","17","18","19","1a","1b","1c","1d","1e","1f",
  "20","21","22","23","24","25","26","27","28","29","2a","2b","2c","2d","2e","2f",
  "30","31","32","33","34","35","36","37","38","39","3a","3b","3c","3d","3e","3f",
  "40","41","42","43","44","45","46","47","48","49","4a","4b","4c","4d","4e","4f",
  "50","51","52","53","54","55","56","57","58","59","5a","5b","5c","5d","5e","5f",
  "60","61","62","63","64","65","66","67","68","69","6a","6b","6c","6d","6e","6f",
  "70","71","72","73","74","75","76","77","78","79","7a","7b","7c","7d","7e","7f",
  "80","81","82","83","84","85","86","87","88","89","8a","8b","8c","8d","8e","8f",
  "90","91","92","93","94","95","96","97","98","99","9a","9b","9c","9d","9e","9f",
  "a0","a1","a2","a3","a4","a5","a6","a7","a8","a9","aa","ab","ac","ad","ae","af",
  "b0","b1","b2","b3","b4","b5","b6","b7","b8","b9","ba","bb","bc","bd","be","bf",
  "c0","c1","c2","c3","c4","c5","c6","c7","c8","c9","ca","cb","cc","cd","ce","cf",
  "d0","d1","d2","d3","d4","d5","d6","d7","d8","d9","da","db","dc","dd","de","df",
  "e0","e1","e2","e3","e4","e5","e6","e7","e8","e9","ea","eb","ec","ed","ee","ef",
  "f0","f1","f2","f3","f4","f5","f6","f7","f8","f9","fa","fb","fc","fd","fe","ff"
};

static uint8_t fromAscii(uint8_t c)
{
  if (c >= '0' && c <= '9')
    return (c - '0');
  if (c >= 'a' && c <= 'f')
    return (c - 'a' + 10);
  if (c >= 'A' && c <= 'F')
    return (c - 'A' + 10);

#if defined(__EXCEPTIONS) || defined(DEBUG)
  throw std::runtime_error("invalid character");
#else
  return 0xff;
#endif
}

static uint8_t ascii_r(uint8_t a, uint8_t b)
{
  return fromAscii(a) * 16 + fromAscii(b);
}

static void HexToBytes(std::string const& hex, uint8_t bytes[])
{
  for (std::string::size_type i = 0, j = 0; i < hex.length(); i += 2, ++j)
  {
    bytes[j] = ascii_r(hex[i], hex[i + 1]);
  }
}


// --------------------------------------------------------------------


// static
std::atomic<uint32_t> CUDASolver::hashes(0u); // statistics only


CUDASolver::CUDASolver() noexcept :
  m_address(ADDRESS_LENGTH),
  m_challenge(UINT256_LENGTH),
  m_target(UINT256_LENGTH),
  m_target_tmp(UINT256_LENGTH),
  m_buffer(ADDRESS_LENGTH + 2 * UINT256_LENGTH),
  m_buffer_tmp(ADDRESS_LENGTH + 2 * UINT256_LENGTH), //this has something to do with updateBuffer
  m_buffer_ready(false),
  m_target_ready(false)
{ }

void CUDASolver::setAddress(std::string const& addr)
{
  cout << "Setting cuda addr \n";

  assert(addr.length() == (ADDRESS_LENGTH * 2 + 2));
  hexToBytes(addr, m_address);
  updateBuffer();
}

void CUDASolver::setChallenge(std::string const& chal)
{
  cout << "Setting cuda chal \n";

  s_challenge = chal;

  assert(chal.length() == (UINT256_LENGTH * 2 + 2));
  hexToBytes(chal, m_challenge);
  updateBuffer();
}

void CUDASolver::setTarget(std::string const& target)
{
  cout << "Setting cuda tar " << target << "\n";

  assert(target.length() <= (UINT256_LENGTH * 2 + 2));
  std::string const t(static_cast<std::string::size_type>(UINT256_LENGTH * 2 + 2) - target.length(), '0');

  s_target = target;

  // Double-buffer system, the trySolution() function will be blocked
  //  only when a change occurs.
  {
    std::lock_guard<std::mutex> g(m_target_mutex);
    hexToBytes("0x" + t + target.substr(2), m_target_tmp);
  }
  m_target_ready = true;
}



// Buffer order: 1-challenge 2-ethAddress 3-solution
void CUDASolver::updateBuffer()
{
  // The idea is to have a double-buffer system in order not to try
  //  to acquire a lock on each hash() loop
  {
    std::lock_guard<std::mutex> g(m_buffer_mutex);
    std::copy(m_challenge.cbegin(), m_challenge.cend(), m_buffer_tmp.begin());
    std::copy(m_address.cbegin(), m_address.cend(), m_buffer_tmp.begin() + m_challenge.size());
  }
  m_buffer_ready = true;
}


//call the sha3.cu init func
void CUDASolver::init()
{
  cout << "CUDA initializing ... \n ";
  gpu_init();
}


 CUDASolver::bytes_t CUDASolver::findSolution( )
{
  cout << "CUDA is trying to find a solution :) \n ";

  hipEventCreate(&start);
  hipEventCreate(&stop);

    char * hash_prefix[50];
   char *s_challenge_chars = s_challenge.c_str();
   char *m_address_chars = m_address.c_str();


  for(int i = 0; i < 32; i++){
       cout <<  s_challenge_chars[i] << "\n";
       hash_prefix[i] = s_challenge_chars[i];
  }

  for(int i = 32; i < 50; i++){
       cout <<  m_address_chars[i-32] << "\n";
       hash_prefix[i] = m_address_chars[i-32];
  }

  unsigned char * s_solution = find_message(s_target.c_str(), hash_prefix );

  CUDASolver::bytes_t byte_solution(32);
  for(int i = 52; i < 84; i++){
    cout << (uint8_t)s_solution[i] << "\n";
    byte_solution[i-52] = (uint8_t)s_solution[i];

  }
  hipEventDestroy(start);
  hipEventDestroy(stop);

  return byte_solution;
}


std::string CUDASolver::hexStr( char* data, int len)
{
    std::stringstream ss;
    ss << std::hex;
    for(int i=0;i<len;++i)
        ss << std::setw(2) << std::setfill('0') << (int)data[i];
    return ss.str();
}


// static
void CUDASolver::hexToBytes(std::string const& hex, bytes_t& bytes)
{

    cout << "hex to bytes: " << hex << "\n";
    cout << bytes.size()  << "\n";
    cout << hex.length()  << "\n";

  assert(hex.length() % 2 == 0);
  assert(bytes.size() == (hex.length() / 2 - 1));
  HexToBytes(hex.substr(2), &bytes[0]);
}

// static
std::string CUDASolver::bytesToString(bytes_t const& buffer)
{
  std::string output;
  output.reserve(buffer.size() * 2 + 1);

  for (unsigned i = 0; i < buffer.size(); ++i)
    output += ascii[buffer[i]];

  return output;
}
